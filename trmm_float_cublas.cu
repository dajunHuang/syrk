#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 20

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    long m = 16384, n = 16384;

    if (argc >= 3) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
    }

    long lda = m, ldb = m, ldc = m;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    float one = 1;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 blocka(16, 16);

    generateUniformMatrixFloat(d_A, lda, m);
    generateUniformMatrixFloat(d_B, ldb, n);

    setInitialValueUpper<float><<<grida, blocka>>>(m, m, d_A, lda, 0);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        PUSH_RANGE("trmm_float_cublas", i);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(start, stream));

        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        POP_RANGE;
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "[cublas strmm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * m * n / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
