#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long m = 16384, n = 16384;

    if (argc >= 3) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
    }

    long lda = m, ldb = m, ldc = m;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    float one = 1;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 blocka(16, 16);

    generateUniformMatrixFloat(d_A, lda, m);
    generateUniformMatrixFloat(d_B, ldb, n);

    setInitialValueUpper<float><<<grida, blocka>>>(m, m, d_A, lda, 0);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "[cublas strmm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * m * n / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
