#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 20

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long m = 16384, n = 16384;

    if (argc >= 3) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
    }

    long lda = m, ldb = m, ldc = m;

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_C = nullptr;

    double one = 1;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * m));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * lda * n));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, m);
    generateUniformMatrixDouble(d_B, ldb, n);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasDtrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        hipblasDtrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C, ldc);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "[cublas dtrmm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * m * n / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
