#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

// C = alpha * A * B + beta * C
// A is m * m col major Lower triangular, B is m * n col major, C is m * n col major 
void trmm(hipblasHandle_t cublasH, int m, int n, float alpha, float *A, int lda,
          float *B, int ldb, float beta, float *C, int ldc, int nb) {
    int num_block = m / nb;
    int left = m % nb;
    float one = 1;
    hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, nb, n, nb, &alpha,
                              A, lda, nb + nb * lda, B, ldb, nb, &beta, C, ldc, nb,
                              num_block);
    if (left > 0) {
        int offset = num_block * nb;
        hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, left, n, left, &alpha,
                    A + offset + offset * lda, lda, B + offset, ldb, &beta,
                    C + offset, ldc);
    }
    for (int i = 1; m / (i * nb) >= 1; i *= 2) {
        num_block = (m - i * nb) / (2 * i * nb);
        left = (m - i * nb) % (2 * i * nb);
        hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, i * nb, n,
                                  i * nb, &alpha, A + i * nb, lda,
                                  2 * (i * nb + i * nb * lda), B, ldb, 2 * i * nb,
                                  &one, C + i * nb, ldc, 2 * i * nb, num_block);
        if (left > 0) {
            left = (left < i * nb) ? (left) : (i * nb);
            int offset_row = i * nb + num_block * (2 * i * nb);
            int offset_col = num_block * (2 * i * nb);
            hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, left, n, i * nb, &alpha,
                        A + offset_row + offset_col * lda, lda, B + offset_col, ldb,
                        &one, C + offset_row, ldc);
        }
    }
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    int m = 16384, n = 16384, nb = 512;
    int check = 0;

    double const fp64_abs_tol = 1.0e-1f;

    if (argc >= 5) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    int lda = m, ldb = m, ldc = m;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    float *d_C_cublas = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C_cublas), sizeof(float) * ldc * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 blocka(16, 16);

    generateUniformMatrixFloat(d_A, lda, m);
    generateUniformMatrixFloat(d_B, ldb, n);

    setInitialValueUpper<float><<<grida, blocka>>>(m, m, d_A, lda, 0);

    CUDA_CHECK(hipDeviceSynchronize());

    trmm(cublasH, m, n, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);

    CUDA_CHECK_LAST_ERROR();

    CUBLAS_CHECK(hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
                             HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda,
                             d_B, ldb, d_C_cublas, ldc));

    CUDA_CHECK(hipDeviceSynchronize());

    dim3 gridc((m + 15) / 16, (n + 15) / 16);
    dim3 blockc(16, 16);
    checkValue<<<gridc, blockc>>>(m, n, d_C, ldc, d_C_cublas, ldc, fp64_abs_tol);

    hipEvent_t start, stop;
    float time1 = 0, time2 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        trmm(cublasH, m, n, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        trmm(cublasH, m, n, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C_cublas,
                    ldc);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        hipblasStrmm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B, ldb, d_C_cublas,
                    ldc);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time2 += temp_time;
    }
    time2 /= NUM_REPEAT;

    if(check) {
        CUDA_CHECK(hipDeviceSynchronize());
        float sonedouble = 1.0, snegonedobule = -1.0;
        hipblasSgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &sonedouble, d_C, ldc,
                    &snegonedobule, d_C_cublas, ldc, d_C, ldc);
        float norm_custom = snorm(m, n, d_C, ldc),
            norm_cublas = snorm(m, n, d_C_cublas, ldc);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
            norm_custom, norm_cublas, norm_custom / norm_cublas);
    }

    std::cout << "[custom strmm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * m * n / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[cublas strmm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time2 << " ms, " << (long)m * m * n / time2 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_C_cublas));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
