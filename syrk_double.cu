#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 20

// C = alpha * A * A^T + beta * C
// A is n * k col major, C is n * n col major
void syrk(hipblasHandle_t cublasH, long n, long k, double alpha, double *A, long lda,
          double beta, double *C, long ldc, long nb) {
    long num_block = n / nb;
    long left = n % nb;
    hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              A, lda, nb, A, lda, nb, &beta, C, ldc, nb + nb * ldc,
                              num_block);
    if (left > 0) {
        long offset = num_block * nb;
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    A + offset, lda, A + offset, lda, &beta,
                    C + offset + offset * ldc, ldc);
    }

    for (long i = 1; i * nb < n; i *= 2) {
        num_block = n / (2 * i * nb);
        left = n - (num_block * 2 * i * nb);
        hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, A + i * nb, lda, 2 * i * nb, A, lda,
                                  2 * i * nb, &beta, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        if (left > i * nb) {
            long offset_row = i * nb + num_block * (2 * i * nb);
            long offset_col = num_block * (2 * i * nb);
            hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, A + offset_row, lda, A + offset_col, lda, &beta,
                        C + offset_row + offset_col * ldc, ldc);
        }
    }
    return;
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long n = 16384, k = 16384, nb = 512;
    int check = 0;

    if (argc >= 5) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    long lda = n, ldc = n;

    double *d_A = nullptr;
    double *d_C = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, k);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        syrk(cublasH, n, k, one, d_A, lda, zero, d_C, ldc, nb);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        syrk(cublasH, n, k, one, d_A, lda, zero, d_C, ldc, nb);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    if (check) {
        double *d_C_cublas = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_cublas),
                              sizeof(double) * ldc * n));
        hipblasDsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                    lda, &zero, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        copy_lower_to_upper(n, d_C, ldc);
        copy_lower_to_upper(n, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &sonedouble, d_C, ldc,
                    &snegonedobule, d_C_cublas, ldc, d_C, ldc);
        double norm_custom = nrm2(cublasH, n, n, d_C, ldc),
               norm_cublas = nrm2(cublasH, n, n, d_C_cublas, ldc);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
               norm_custom, norm_cublas, norm_custom / norm_cublas);
        CUDA_CHECK(hipFree(d_C_cublas));
    }

    std::cout << "[custom dsyrk] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, " << (long)n * n * k / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
