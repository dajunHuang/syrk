#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

// C = alpha * A * A^T + beta * C
// A is n * k col major, C is n * n col major
void syrk(hipblasHandle_t cublasH, int n, int k, double alpha, double *A, int lda,
          double beta, double *C, int ldc, int nb) {
    int num_block = n / nb;
    int left = n % nb;
    hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              A, lda, nb, A, lda, nb, &beta, C, ldc, nb + nb * ldc,
                              num_block);
    if (left > 0) {
        int offset = num_block * nb;
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    A + offset, lda, A + offset, lda, &beta,
                    C + offset + offset * ldc, ldc);
    }

    for (int i = 1; i * nb < n; i *= 2) {
        num_block = n / (2 * i * nb);
        left = n - (num_block * 2 * i * nb);
        hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, A + i * nb, lda, 2 * i * nb, A, lda,
                                  2 * i * nb, &beta, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        if (left > i * nb) {
            int offset_row = i * nb + num_block * (2 * i * nb);
            int offset_col = num_block * (2 * i * nb);
            hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, A + offset_row, lda, A + offset_col, lda, &beta,
                        C + offset_row + offset_col * ldc, ldc);
        }
    }
    return;
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    int n = 16384, k = 16384, nb = 512;
    int check = 0;

    if (argc >= 5) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    int lda = n, ldc = n;

    double *d_A = nullptr;
    double *d_C = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, k);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        syrk(cublasH, n, k, one, d_A, lda, zero, d_C, ldc, nb);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        syrk(cublasH, n, k, one, d_A, lda, zero, d_C, ldc, nb);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    if (check) {
        double *d_C_cublas = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_cublas),
                              sizeof(double) * ldc * n));
        hipblasDsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                    lda, &zero, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        copy_lower_to_upper(n, d_C, ldc);
        copy_lower_to_upper(n, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &sonedouble, d_C, ldc,
                    &snegonedobule, d_C_cublas, ldc, d_C, ldc);
        double norm_custom = snorm(n, n, d_C, ldc),
               norm_cublas = snorm(n, n, d_C_cublas, ldc);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
               norm_custom, norm_cublas, norm_custom / norm_cublas);
        CUDA_CHECK(hipFree(d_C_cublas));
    }

    std::cout << "[custom dsyrk] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, " << (long)n * n * k / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
