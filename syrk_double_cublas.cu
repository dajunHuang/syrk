#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    int n = 16384, k = 16384;

    if (argc >= 3) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
    }

    int lda = n, ldc = n;

    double *d_A = nullptr;
    double *d_C = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, k);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasDsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one,
            d_A, lda, &zero, d_C, ldc);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        hipblasDsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one,
            d_A, lda, &zero, d_C, ldc);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "[cublas dsyrk] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, " << (long)n * n * k / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
