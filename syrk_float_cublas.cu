#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    int n = 16384, k = 16384;

    if (argc >= 3) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
    }

    int lda = n, ldc = n;

    float *d_A = nullptr;
    float *d_C = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));

    generateUniformMatrixFloat(d_A, lda, k);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasSsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                    lda, &zero, d_C, ldc);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        hipblasSsyrk(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                    lda, &zero, d_C, ldc);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "[cublas ssyrk] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, " << (long)n * n * k / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
