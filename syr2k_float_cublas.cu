#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 20

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    long n = 16384, k = 16384;

    if (argc >= 3) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
    }

    long lda = n, ldb = n, ldc = n;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));

    generateUniformMatrixFloat(d_A, lda, k);
    generateUniformMatrixFloat(d_B, ldb, k);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        hipblasSsyr2k(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one,
            d_A, lda, d_B, ldb, &zero, d_C, ldc);
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        PUSH_RANGE("syr2k_float_cublas", i);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(start, stream));

        hipblasSsyr2k(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one,
            d_A, lda, d_B, ldb, &zero, d_C, ldc);

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        POP_RANGE;
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "[cublas ssyr2k] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, "
              << ((long)n * k * n * 2) / time1 / 1e9 << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
