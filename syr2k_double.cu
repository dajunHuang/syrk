#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 1
#define NUM_REPEAT 2

// C = alpha * A * B^T + alpha * B * A^T + beta * C
// A is n * k col major, B is n * k col major, C is n * n col major
void syr2k(hipblasHandle_t cublasH, long n, long k, double alpha, double *A, long lda,
           double *B, long ldb, double beta, double *C, long ldc, long nb) {
    double one = 1;
    long num_block = n / nb;
    long left = n % nb;
    hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              A, lda, nb, B, ldb, nb, &beta, C, ldc, nb + nb * ldc,
                              num_block);
    hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              B, ldb, nb, A, lda, nb, &one, C, ldc, nb + nb * ldc,
                              num_block);
    if (left > 0) {
        long offset = num_block * nb;
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    A + offset, lda, B + offset, ldb, &beta,
                    C + offset + offset * ldc, ldc);
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    B + offset, ldb, A + offset, lda, &one,
                    C + offset + offset * ldc, ldc);
    }

    for (long i = 1; i * nb < n; i *= 2) {
        num_block = n / (2 * i * nb);
        left = n - (num_block * 2 * i * nb);
        hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, A + i * nb, lda, 2 * i * nb, B, ldb,
                                  2 * i * nb, &beta, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        hipblasDgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, B + i * nb, ldb, 2 * i * nb, A, lda,
                                  2 * i * nb, &one, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        if (left > i * nb) {
            long offset_row = i * nb + num_block * (2 * i * nb);
            long offset_col = num_block * (2 * i * nb);
            hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, A + offset_row, lda, B + offset_col, ldb, &beta,
                        C + offset_row + offset_col * ldc, ldc);
            hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, B + offset_row, ldb, A + offset_col, lda, &one,
                        C + offset_row + offset_col * ldc, ldc);
        }
    }
    return;
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long n = 16384, k = 16384, nb = 512;
    int check = 0;

    if (argc >= 5) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    long lda = n, ldb = n, ldc = n;

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_C = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * lda * k));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * ldc * n));

    generateUniformMatrixDouble(d_A, lda, k);
    generateUniformMatrixDouble(d_B, ldb, k);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        syr2k(cublasH, n, k, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i{0}; i < NUM_REPEAT; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        syr2k(cublasH, n, k, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipDeviceSynchronize());

    if (check) {
        double *d_C_cublas = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_cublas),
                              sizeof(double) * ldc * n));
        hipblasDsyr2k(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                     lda, d_B, ldb, &zero, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        copy_lower_to_upper(n, d_C, ldc);
        copy_lower_to_upper(n, d_C_cublas, ldc);
        CUDA_CHECK(hipDeviceSynchronize());
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &sonedouble, d_C, ldc,
                    &snegonedobule, d_C_cublas, ldc, d_C, ldc);
        double norm_custom = snorm(n, n, d_C, ldc),
               norm_cublas = snorm(n, n, d_C_cublas, ldc);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
               norm_custom, norm_cublas, norm_custom / norm_cublas);
        CUDA_CHECK(hipFree(d_C_cublas));
    }

    std::cout << "[custom dsyr2k] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, "
              << ((long)n * k * n * 2) / time1 / 1e9 << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
