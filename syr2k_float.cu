#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

#define NUM_WARPUP 5
#define NUM_REPEAT 20

// C = alpha * A * B^T + alpha * B * A^T + beta * C
// A is n * k col major, B is n * k col major, C is n * n col major
void syr2k(hipblasHandle_t cublasH, long n, long k, float alpha, float *A, long lda,
           float *B, long ldb, float beta, float *C, long ldc, long nb) {
    float one = 1;
    long num_block = n / nb;
    long left = n % nb;
    hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              A, lda, nb, B, ldb, nb, &beta, C, ldc, nb + nb * ldc,
                              num_block);
    hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, nb, k, &alpha,
                              B, ldb, nb, A, lda, nb, &one, C, ldc, nb + nb * ldc,
                              num_block);
    if (left > 0) {
        long offset = num_block * nb;
        hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    A + offset, lda, B + offset, ldb, &beta,
                    C + offset + offset * ldc, ldc);
        hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left, left, k, &alpha,
                    B + offset, ldb, A + offset, lda, &one,
                    C + offset + offset * ldc, ldc);
    }

    for (long i = 1; i * nb < n; i *= 2) {
        num_block = n / (2 * i * nb);
        left = n - (num_block * 2 * i * nb);
        hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, A + i * nb, lda, 2 * i * nb, B, ldb,
                                  2 * i * nb, &beta, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        hipblasSgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, i * nb, i * nb,
                                  k, &alpha, B + i * nb, ldb, 2 * i * nb, A, lda,
                                  2 * i * nb, &one, C + i * nb, ldc,
                                  2 * (i * nb + i * nb * ldc), num_block);
        if (left > i * nb) {
            long offset_row = i * nb + num_block * (2 * i * nb);
            long offset_col = num_block * (2 * i * nb);
            hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, A + offset_row, lda, B + offset_col, ldb, &beta,
                        C + offset_row + offset_col * ldc, ldc);
            hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, left - i * nb, i * nb, k,
                        &alpha, B + offset_row, ldb, A + offset_col, lda, &one,
                        C + offset_row + offset_col * ldc, ldc);
        }
    }
    return;
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    long n = 16384, k = 16384, nb = 512;
    int check = 0;

    if (argc >= 5) {
        n = atoi(argv[1]);
        k = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    long lda = n, ldb = n, ldc = n;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * ldc * n));

    generateUniformMatrixFloat(d_A, lda, k);
    generateUniformMatrixFloat(d_B, ldb, k);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    for (int i{0}; i < NUM_WARPUP; ++i) {
        syr2k(cublasH, n, k, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);
    }
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i{0}; i < NUM_REPEAT; ++i) {
        PUSH_RANGE("syr2k_float", i);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(start, stream));

        syr2k(cublasH, n, k, one, d_A, lda, d_B, ldb, zero, d_C, ldc, nb);

        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipEventRecord(stop, stream));
        CUDA_CHECK(hipEventSynchronize(stop));
        POP_RANGE;
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
        time1 += temp_time;
    }
    time1 /= NUM_REPEAT;

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (check) {
        float *d_C_cublas = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_cublas),
                              sizeof(float) * ldc * n));
        hipblasSsyr2k(cublasH, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &one, d_A,
                     lda, d_B, ldb, &zero, d_C_cublas, ldc);
        CUDA_CHECK(hipStreamSynchronize(stream));
        copy_lower_to_upper(n, d_C, ldc);
        copy_lower_to_upper(n, d_C_cublas, ldc);
        CUDA_CHECK(hipStreamSynchronize(stream));
        float sonedouble = 1.0, snegonedobule = -1.0;
        hipblasSgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &sonedouble, d_C, ldc,
                    &snegonedobule, d_C_cublas, ldc, d_C, ldc);
        float norm_custom = nrm2(cublasH, n, n, d_C, ldc),
              norm_cublas = nrm2(cublasH, n, n, d_C_cublas, ldc);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
               norm_custom, norm_cublas, norm_custom / norm_cublas);
        CUDA_CHECK(hipFree(d_C_cublas));
    }

    std::cout << "[custom ssyr2k] " << "m: " << n << ", n: " << k << ", "
              << "latency: " << time1 << " ms, "
              << ((long)n * k * n * 2) / time1 / 1e9 << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
