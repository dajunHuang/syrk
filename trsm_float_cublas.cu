#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long m = 16384, n = 16384;

    if (argc >= 3) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
    }

    long lda = m, ldb = m;

    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_B_custom = nullptr;

    float one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * lda * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(float) * lda * n));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B_custom), sizeof(float) * ldb * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 gridb((m + 15) / 16, (n + 15) / 16);
    dim3 block(16, 16);

    setInitialValue<<<grida, block>>>(m, m, d_A, lda, one);
    setInitialValueUpper<<<grida, block>>>(m, m, d_A, lda, zero);

    generateUniformMatrixFloat(d_B, ldb, n);

    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    hipblasStrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B_custom, ldb);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    hipblasStrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B_custom, ldb);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    time1 += temp_time;

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "[cublas strsm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * n * n / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_B_custom));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
