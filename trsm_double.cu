#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

// A * X = alpha * B
// A is m * m col major Lower triangular, B is m * n col major, overwrited by X
void trsm(hipblasHandle_t cublasH, long m, long n, double alpha, double *A, long lda,
          double *B, long ldb, long nb) {
    double sonedouble = 1.0, snegonedobule = -1.0;
    if (m <= nb) {
        CUBLAS_CHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
                                 HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha, A,
                                 lda, B, ldb));
        return;
    }

    trsm(cublasH, m / 2, n, alpha, A, lda, B, ldb, nb);

    long left = m - m / 2;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, left, n, m / 2,
                             &snegonedobule, A + m / 2, lda, B, ldb, &sonedouble,
                             B + m / 2, ldb));

    trsm(cublasH, left, n, alpha, A + m / 2 + m / 2 * lda, lda, B + m / 2, ldb, nb);
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    long m = 16384, n = 16384, nb = 512;
    int check = 0;

    // double const fp64_abs_tol = 1.0e-4f;

    if (argc >= 5) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        nb = atoi(argv[3]);
        check = atoi(argv[4]);
    }

    long lda = m, ldb = m;

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_B_custom = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * m));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_custom),
                          sizeof(double) * ldb * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 gridb((m + 15) / 16, (n + 15) / 16);
    dim3 block(16, 16);

    setInitialValue<<<grida, block>>>(m, m, d_A, lda, one);
    setInitialValueUpper<<<grida, block>>>(m, m, d_A, lda, zero);

    generateUniformMatrixDouble(d_B, ldb, n);

    hipEvent_t start, stop;
    float time1 = 0, temp_time = 0;

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    trsm(cublasH, m, n, one, d_A, lda, d_B_custom, ldb, nb);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    trsm(cublasH, m, n, one, d_A, lda, d_B_custom, ldb, nb);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    time1 += temp_time;

    CUDA_CHECK(hipDeviceSynchronize());

    if (check) {
        double *d_B_cublas = nullptr;
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_cublas),
                              sizeof(double) * ldb * n));
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(d_B_cublas, d_B, ldb * n, hipMemcpyDeviceToDevice));
        hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B_cublas, ldb);
        CUDA_CHECK(hipDeviceSynchronize());
        double sonedouble = 1.0, snegonedobule = -1.0;
        hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &sonedouble, d_B_custom,
                    ldb, &snegonedobule, d_B_cublas, ldb, d_B_custom, ldb);
        double norm_custom = nrm2(cublasH, m, n, d_B_custom, ldb),
               norm_cublas = nrm2(cublasH, m, n, d_B_cublas, ldb);
        printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
               norm_custom, norm_cublas, norm_custom / norm_cublas);
        CUDA_CHECK(hipFree(d_B_cublas));
    }

    std::cout << "[custom dtrsm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, " << (long)m * m * n / 2 / time1 / 1e9
              << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_B_custom));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
