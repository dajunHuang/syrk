#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "utils.h"

// A * X = alpha * B
// A is m * m col major Lower triangular, B is m * n col major, overwrited by X
void trsm(hipblasHandle_t cublasH, int m, int n, double alpha, double *A, int lda, double *B,
          int ldb, int nb) {
    double sonedouble = 1.0, snegonedobule = -1.0;
    if (m <= nb) {
        CUBLAS_CHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
                                 HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n,
                                 &alpha, A, lda, B, ldb));
        return;
    }

    trsm(cublasH, m / 2, n, alpha, A, lda, B, ldb, nb);

    int left = m - m / 2;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, left, n, m / 2,
                             &snegonedobule, A + m / 2, lda, B, ldb,
                             &sonedouble, B + m / 2, ldb));


    trsm(cublasH, left, n, alpha, A + m / 2 + m / 2 * lda, lda, B + m / 2, ldb, nb);
}

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;

    int m = 16384, n = 16384, nb = 512;

    double const fp64_abs_tol = 1.0e-4f;

    if (argc >= 4) {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        nb = atoi(argv[3]);
    }

    int lda = m, ldb = m;

    // assert(m % nb == 0);

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_B_custom = nullptr;
    double *d_B_cublas = nullptr;

    double one = 1, zero = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: copy A to device */
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * m));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * lda * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_custom),
                          sizeof(double) * ldb * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_cublas),
                          sizeof(double) * ldb * n));

    dim3 grida((m + 15) / 16, (m + 15) / 16);
    dim3 gridb((m + 15) / 16, (n + 15) / 16);
    dim3 block(16, 16);

    setInitialValue<<<grida, block>>>(m, m, d_A, lda, one);
    setInitialValueUpper<<<grida, block>>>(m, m, d_A, lda, zero);

    generateUniformMatrixDouble(d_B, ldb, n);

    // print_device_matrix(d_A, lda, 32, 32);

    CUDA_CHECK(hipDeviceSynchronize());


    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_B_cublas, d_B, ldb * n, hipMemcpyDeviceToDevice));

    CUDA_CHECK(hipDeviceSynchronize());

    trsm(cublasH, m, n, one, d_A, lda, d_B_custom, ldb, nb);

    CUDA_CHECK(hipDeviceSynchronize());

    CUBLAS_CHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
                             HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda,
                             d_B_cublas, ldb));
    CUDA_CHECK(hipDeviceSynchronize());

    checkValue<<<gridb, block>>>(m, n, d_B_custom, ldb, d_B_cublas, ldb,
                                  fp64_abs_tol);

    hipEvent_t start, stop;
    float time1 = 0, time2 = 0, temp_time = 0;

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    trsm(cublasH, m, n, one, d_A, lda, d_B_custom, ldb, nb);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(d_B_custom, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    trsm(cublasH, m, n, one, d_A, lda, d_B_custom, ldb, nb);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    time1 += temp_time;

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(d_B_cublas, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B_cublas, ldb);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(d_B_cublas, d_B, ldb * n, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, &one, d_A, lda, d_B_cublas, ldb);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_LAST_ERROR();
    CUDA_CHECK(hipEventElapsedTime(&temp_time, start, stop));
    time2 += temp_time;

    CUDA_CHECK(hipDeviceSynchronize());
    double sonedouble = 1.0, snegonedobule = -1.0;
    hipblasDgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &sonedouble, d_B_custom, ldb,
                &snegonedobule, d_B_cublas, ldb, d_B_custom, ldb);
    double norm_custom = snorm(m, n, d_B_custom, ldb),
          norm_cublas = snorm(m, n, d_B_cublas, ldb);
    printf("norm_custom: %.6e, norm_cublas: %.6e, forward error: %.6e\n",
           norm_custom, norm_cublas, norm_custom / norm_cublas);

    std::cout << "[custom dtrsm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time1 << " ms, "
              << (long)m * n * n / time1 / 1e9 << " TFLOPS" << std::endl;
    std::cout << "[cublas dtrsm] " << "m: " << m << ", n: " << n << ", "
              << "latency: " << time2 << " ms, "
              << (long)m * n * n / time2 / 1e9 << " TFLOPS" << std::endl;
    std::cout << "[Free memory] " << free_mem() / 1024 / 1024 / 1024 << " GB"
              << std::endl;

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
